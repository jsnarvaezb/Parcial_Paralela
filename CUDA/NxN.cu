#include "hip/hip_runtime.h"
/*
 *  file name: matrix.cu
 *
 *  matrix.cu contains the code that realize some common used matrix operations in CUDA
 *
 *  this is a toy program for learning CUDA, some functions are reusable in other project
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCK_SIZE 16



/*
*********************************************************************
function name: gpu_square_matrix_mult
description: dot product of two matrix (not only square) in GPU
parameters:
            &a GPU device pointer to a n X n matrix (A)
            &b GPU device pointer to a n X n matrix (B)
            &c GPU device output purpose pointer to a n X n matrix (C)
            to store the result
Note:
    grid and block should be configured as:
        dim3 dim_grid((n - 1) / BLOCK_SIZE + 1, (n - 1) / BLOCK_SIZE + 1, 1);
        dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);
return: none
*********************************************************************
*/
__global__ void gpu_square_matrix_mult(int *d_a, int *d_b, int *d_result, int n)
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub)
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        if(idx >= n*n)
        {
            // n may not divisible by BLOCK_SIZE
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        if(idx >= n*n)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int n = 0; n < BLOCK_SIZE; ++n)
        {
            tmp += tile_a[threadIdx.y][n] * tile_b[n][threadIdx.x];
        }
        __syncthreads();
    }
    if(row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}

/*
*********************************************************************
function name: main
description: test and compare
parameters:
            none
return: none
*********************************************************************
*/
int main(int argc, char const *argv[])
{
    int n;
    /* Fixed seed for illustration */
    //srand(3333);

    int n = atoi(argv[1]);


    int *h_a, *h_b, *h_c;
    hipHostMalloc((void **) &h_a, sizeof(int)*n*n);
    hipHostMalloc((void **) &h_b, sizeof(int)*n*n);
    hipHostMalloc((void **) &h_c, sizeof(int)*n*n);

    // random initialize matrix A
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = rand() % 1024;
        }
    }

    // random initialize matrix B
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            h_b[i * n + j] = rand() % 1024;
        }
    }

    // Allocate memory space on the device
    int *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(int)*n*n);
    hipMalloc((void **) &d_b, sizeof(int)*n*n);
    hipMalloc((void **) &d_c, sizeof(int)*n*n);

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, h_a, sizeof(int)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*n*n, hipMemcpyHostToDevice);

    unsigned int grid_rows = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    // Launch kernel

    gpu_square_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);

    // Transefr results from device to host
    hipMemcpy(h_c, d_c, sizeof(int)*n*n, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();


    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    return 0;
}
